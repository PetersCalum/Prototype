#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <ctype.h>
#include <time.h>
#include <Windows.h>
#include <hip/hip_runtime.h> //necessary

#define IMAGE_WIDTH		5
#define IMAGE_HEIGHT	7
#define PATTERN_SIZE	IMAGE_WIDTH*IMAGE_HEIGHT
#define NETWORK_SIZE	PATTERN_SIZE*PATTERN_SIZE

void
draw_pattern(int* pattern);

__global__ void
parallel_train_network(double* network, int* formatted_image) {
	//again, assumes a [-1, 1] formatted pattern
	//rather than run that in device code, supply a preformatted
	//image instead.
	*(network + threadIdx.x + blockIdx.x * blockDim.x) += formatted_image[threadIdx.x] * formatted_image[blockIdx.x];
	//In this instance, the block variables represent the inner loop
	//because the network is set up such that the inner loop controls the 
	//contiguous storage.
}

__global__ void
parallel_recall_step(double* network, int* pattern) {
	//unlike the serial recall step, this processes the entire image at once.
	//code-wise, it's more similar to neuron_out.

	//makes things easier to write
	int thread_id = threadIdx.x;

	//split off each pixel's region of the outputs into shared memory
	__shared__ double summed_data[PATTERN_SIZE];

	//calculate the sum normally, but store them in the shared array
	//blockIdx is the pixel being summed (AKA neuron in)
	*(summed_data + thread_id) = *(pattern + thread_id) * *(network + blockIdx.x + thread_id * blockDim.x);

	__syncthreads(); //threads shouldn't move beyond here until the array is full

					 //a more efficient loop could be implemented if the image size was a power of two.
					 //would cause even more serious divergence between parallel and serial implemenations
	for (int i = 1; i < blockDim.x; i *= 2) {
		if (thread_id % (2 * i) == 0)
			if (thread_id + i < blockDim.x) //would be safe without this if a power of two
				summed_data[thread_id] += summed_data[thread_id + i];
		__syncthreads();
	}


	if (thread_id == 0) {
		//0th element stores the final sum
		if (summed_data[thread_id] > 0)
			pattern[blockIdx.x] = 1;
		else
			pattern[blockIdx.x] = -1;
	}
}

/*HOST CODE FOLLOWS*/

void
parallel_recall(double* network, int* pattern) {
	bool exit = false;
	int counter = 0;
	do {
		//declare the device-specific variables
		double* device_network;
		int* device_pattern;
		//allocate the device memory for the variables.
		hipMalloc((void **)&device_network, sizeof(network));
		hipMalloc((void **)&device_pattern, sizeof(pattern));

		//copy the data over
		hipMemcpy(device_network, network, sizeof(device_network), hipMemcpyHostToDevice);
		hipMemcpy(device_pattern, pattern, sizeof(device_pattern), hipMemcpyHostToDevice);

		//launch kernel on the GPU
		parallel_recall_step<<<PATTERN_SIZE, PATTERN_SIZE>>>(device_network, device_pattern); //run with as many blocks as the pattern size, and that many threads per block.

		//hipDeviceSynchronize(); 

		//copy updated pattern back from the device and free device memory
		hipMemcpy(pattern, device_pattern, sizeof(pattern), hipMemcpyDeviceToHost);
		hipFree(device_network);
		hipFree(device_pattern);

		//draw pattern
		draw_pattern(pattern);

		char option;
		if (counter == 50) {
			printf("\n Enter Q to quit or anything else to continue.");
			Sleep(500);
			scanf("%c", &option);
			option = toupper(option);
			if (option == 'Q')
				exit = true;
			counter = 0;
		}
		counter++;
		Sleep(100);
	} while (exit == false);
}


void
negative_image(int* image) {
	for (int i = 0; i < PATTERN_SIZE; i++) {
		//convert 0's to negatives
		*(image + i) = *(image + i) * 2 - 1; //1*2 = 2, 0*2 = 0
	}
}

void
parallel_train_starter(double* network, int* image) {
	//converts the image to be learned into a [-1, 1] format
	//new array to not change the original pattern
	int transformed_image[PATTERN_SIZE];
	for (int i = 0; i < PATTERN_SIZE; i++) {
		transformed_image[i] = *(image + i);
	}

	//declaration and memory allocation
	double* device_network;
	int* device_image;
	hipMalloc((void **)&device_network, sizeof(network));
	hipMalloc((void **)&device_image, sizeof(transformed_image));

	hipMemcpy(device_network, network, sizeof(device_network), hipMemcpyHostToDevice);
	hipMemcpy(device_image, image, sizeof(device_image), hipMemcpyHostToDevice);

	parallel_train_network<<<PATTERN_SIZE, PATTERN_SIZE>>>(device_network, device_image);

	hipMemcpy(network, device_network, sizeof(network), hipMemcpyDeviceToHost);

	hipFree(device_network);
	hipFree(device_image);
		 
}

void
train_network(double* network, int* image) {
	//converts the image to be learned into a [-1, 1] format
	//new array to not change the original pattern
	int transformed_image[PATTERN_SIZE];
	for (int i = 0; i < PATTERN_SIZE; i++) {
		transformed_image[i] = *(image + i);
	}
	negative_image(transformed_image);

	for (int i = 0; i < PATTERN_SIZE; i++) { //relate every pixel
		for (int j = 0; j < PATTERN_SIZE; j++) { //to every other pixel, including itself 
			*(network + i + j*PATTERN_SIZE) += transformed_image[i] * transformed_image[j];
			//note that this stores the correlations of every i to one j continuously--e.g., all the relations TO
			//the first pixel are stored from 0 to 34.
		}
	}
}

void
draw_pattern(int* pattern) {
	system("cls"); //only works on Windows
	for (int i = 0; i < PATTERN_SIZE; i++) {
		if (i % 5 == 0)
			printf("\n");
		if (*(pattern + i) == 1)
			printf("#");
		else
			printf(".");
	}
}

void
generate_cue(int* pattern, int* cue_pattern, int noise_threshold) {
	for (int i = 0; i < PATTERN_SIZE; i++) {
		cue_pattern[i] = pattern[i];
	}
	for (int i = 0; i < PATTERN_SIZE; i++) {
		//flip pixel according to noise
		if (rand() % 101 < noise_threshold)
			cue_pattern[i] = 1 - cue_pattern[i];
	}
}

int
neuron_out(double* network, int* pattern, int neuron_in) {
	double sum_weights = 0;
	int pixel_value = 0;
	for (int i = 0; i < PATTERN_SIZE; i++) {
		sum_weights += *(pattern + i) * *(network + neuron_in + i*PATTERN_SIZE);
		//multiply the value of the ith pixel by the correlation of i and the current pixel
		//add to the weight. The sign of the weight shows wheteher the pixel should be on.
	}
	if (sum_weights > 0)
		pixel_value = 1;
	else
		pixel_value = -1;
	return pixel_value;
}

void
shuffle(int *array, size_t n) {
	if (n > 1)
	{
		size_t i;
		for (i = 0; i < n - 1; i++)
		{
			size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
			int t = array[j];
			array[j] = array[i];
			array[i] = t;
		}
	}
}

void
recall_step(double* network, int* pattern, int* sequence, int step) {
	if (step == 0)
		shuffle(sequence, sizeof(sequence)); //randomise sequence order
											 //rather than consistently evaluating from the first pixel
											 //then update the image based on the network
	int new_pixel;
	new_pixel = neuron_out(network, pattern, sequence[step]); //see whether the network's
															  //correlations suggest a pixel should be different.
	if (new_pixel != pattern[sequence[step]])
		pattern[sequence[step]] = new_pixel;
	draw_pattern(pattern);
}

void
recall(double* network, int* pattern) {
	int sequence[PATTERN_SIZE];
	for (int i = 0; i < PATTERN_SIZE; i++) {
		sequence[i] = i;
	}
	int step = 0;
	bool exit = false;
	do {
		recall_step(network, pattern, sequence, step);
		step++;
		if (step == PATTERN_SIZE) {
			step = 0; //causes the next recall step to shuffle the order once more.
			char option;
			printf("\n Enter Q to quit or anything else to continue.");
			Sleep(500);
			scanf("%c", &option);
			option = toupper(option);
			if (option == 'Q')
				exit = true;
		}
		Sleep(100);
	} while (exit == false);
}

int main(void) {

	//start RNG
	srand(time(NULL));

	//array pattern for the letter A
	int a_pattern[] = { 0, 0, 1, 0, 0,
		0, 1, 0, 1, 0,
		1, 0, 0, 0, 1,
		1, 1, 1, 1, 1,
		1, 0, 0, 0, 1,
		1, 0, 0, 0, 1,
		1, 0, 0, 0, 1 };

	//array pattern for the letter s
	int s_pattern[] = { 1, 1, 1, 1, 1,
		1, 0, 0, 0, 0,
		0, 1, 0, 0, 0,
		0, 0, 1, 0, 0,
		0, 0, 0, 1, 0,
		0, 0, 0, 0, 1,
		1, 1, 1, 1, 1 };

	//array pattern for the letter t
	int t_pattern[] = { 1, 1, 1, 1, 1,
		0, 0, 1, 0, 0,
		0, 0, 1, 0, 0,
		0, 0, 1, 0, 0,
		0, 0, 1, 0, 0,
		0, 0, 1, 0, 0,
		0, 0, 1, 0, 0 };

	//array pattern for the letter u
	int u_pattern[] = { 1, 0, 0, 0, 1,
		1, 0, 0, 0, 1,
		1, 0, 0, 0, 1,
		1, 0, 0, 0, 1,
		1, 0, 0, 0, 1,
		1, 0, 0, 0, 1,
		1, 1, 1, 1, 1 };

	//create a blank network
	double weighted_network[NETWORK_SIZE];
	for (int i = 0; i < NETWORK_SIZE; i++) {
		weighted_network[i] = 0;
	}

	printf("Enter G to train using the GPU instead. \n");
	char gpu_train;
	gpu_train = scanf("%s", &gpu_train);
	gpu_train = toupper(gpu_train);
	if (gpu_train == 'G') {
		parallel_train_starter(weighted_network, a_pattern);
		parallel_train_starter(weighted_network, s_pattern);
		parallel_train_starter(weighted_network, t_pattern);
		parallel_train_starter(weighted_network, u_pattern);
	}
	else {
		train_network(weighted_network, a_pattern);
		train_network(weighted_network, s_pattern);
		train_network(weighted_network, t_pattern);
		train_network(weighted_network, u_pattern);
	}
	int cue[PATTERN_SIZE];

	bool exit = false;
	do {
		bool generated = false;

		char gpu;
		char option;
		int noise;
		printf("Enter G to use the GPU instead. \n");
		scanf("%s", &gpu);
		gpu = toupper(gpu);
		printf("Enter a character: A, S, T, U to recall that letter, Q to quit. \n");
		scanf("%s", &option);
		option = toupper(option);
		if (option != 'Q') {
			printf("Please enter a value from 0-100 for noise. \n");
			scanf("%d", &noise);
		}
		switch (option) {
		case 'A':
			generate_cue(a_pattern, cue, noise);
			generated = true;
			break;
		case 'S':
			generate_cue(s_pattern, cue, noise);
			generated = true;
			break;
		case 'T':
			generate_cue(t_pattern, cue, noise);
			generated = true;
			break;
		case 'U':
			generate_cue(u_pattern, cue, noise);
			generated = true;
			break;
		case 'Q':
			exit = true;
		}
		if (generated) {
			negative_image(cue);
			draw_pattern(cue);
			if (gpu == 'G')
				parallel_recall(weighted_network, cue);
			else
				recall(weighted_network, cue);
		}
	} while (exit == false);

	return 0;
}